
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<math.h>
#include <time.h>
#include <omp.h>
#define N 10000000
//#define N 30000000
//#define N 50000000

__global__ void vecKernel(float*x, float *y, int num) {
	// parallel control via varying index
	int my_i = threadIdx.x + blockIdx.x*blockDim.x;
	// handle my_i exceeds num
 if(my_i<num){
	y[my_i] = 1 / exp((x[my_i] - 2)*(x[my_i] - 2)) 
            + 1 / exp((x[my_i] - 6)*(x[my_i] - 6) / 10) 
            + 1 / (x[my_i] * x[my_i] + 1);
	}	
}

int main()
{
	float *x, *y;
  float max=0;
	int i;
  int index;
  float len;
  len = 200.0/N;
  
  // device timer
  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop); 
  
  clock_t start_cpu, finish1_cpu,finish2_cpu,finish3_cpu,finish4_cpu,start_serial,stop_serial;
  start_cpu = clock();
  
  //memory for device(GPU)
	float *x_dev, *y_dev;
	//set up array;
	x = (float *)malloc(N * sizeof(*x));
	y = (float *)malloc(N * sizeof(*y));
	//set up device memory
	hipMalloc(&x_dev, N * sizeof(*x));
	hipMalloc(&y_dev, N * sizeof(*y));
  
  finish1_cpu = clock();
  // init vars
#pragma omp parallel for 
	      for (i = 0; i < N ; i++)
	      {
		       x[i] = (-100)+len*i; 
	      }
  // copy to device
	hipMemcpy(x_dev, x, N * sizeof(float), hipMemcpyHostToDevice);
  // see if any errors
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess) {
      printf("(1) CUDA RT error: %s \n",hipGetErrorString(err));
  }
  finish2_cpu = clock();

	// init the kernel
	int threadsPerBlock;
  float maxThreadsPerBlock = 1024;
	threadsPerBlock = (N > maxThreadsPerBlock ? maxThreadsPerBlock : N);
	int blocks = ceil((float)N / (float)threadsPerBlock);                           // ensure cover all elements of x
	printf("eg %d threads/block on %d blocks\n", threadsPerBlock, blocks);

	hipEventRecord(start,0);
  vecKernel <<<blocks, threadsPerBlock >>> (x_dev,y_dev, N);
  // copy results back  
	hipMemcpy(y, y_dev, N * sizeof(float), hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  finish3_cpu = clock();
  
  // see if any errors 
   err = hipGetLastError();
   if ( err != hipSuccess) {
      printf("(2) CUDA RT error: %s \n",hipGetErrorString(err));
   }
  
  #pragma omp parallel for
	for (i = 0; i < N ; i++)
	{
		if (max < y[i]) {
			max = y[i];
			index = i;
		}
	}
	printf("CUDA: Max is %dth number = %.4f\n",index,max);
	
	/* --- CLEAN UP GPU --- */
  // release device memory
	hipFree(x_dev);    hipFree(y_dev);  
  finish4_cpu = clock();  
  
  //serial code
  start_serial = clock();
  for (i=0;i<N;i++)
  {
      x[i] = (-100)+len*i;
  }
  
  for (i=0;i<N;i++)
  { 
      y[i]= 1 / exp((x[i] - 2)*(x[i] - 2)) + 1 / exp((x[i] - 6)*(x[i] - 6) / 10) + 1 / (x[i] * x[i] + 1);
  
  }
  for (i = 0; i < N ; i++)
	{
		if (max < y[i]) {
			max = y[i];
			index = i;
		}
	}
  printf("Serial: Max is %dth number = %.4f\n",index,max);
  stop_serial = clock();
  
  // determine GPU time
  float eTime;
  hipEventElapsedTime(&eTime, start, stop);
  printf("elapsed GPU time: %f millisecs\n", eTime);
  printf("elapsed CPU time for init+cpy:   %f secs\n", (float)(finish2_cpu-finish1_cpu)/CLOCKS_PER_SEC);
  printf("elapsed CPU time for kernel:   %f secs\n", (float)(finish3_cpu-finish2_cpu)/CLOCKS_PER_SEC);
  printf("elapsed CPU time for finding Max:   %f secs\n", (float)(finish4_cpu-finish3_cpu)/CLOCKS_PER_SEC);
  printf("elapsed CPU time *total*:   %f secs\n", (float)(finish4_cpu-start_cpu)/CLOCKS_PER_SEC);
  printf("Serial Code: %.3f seconds\n", (float)(stop_serial-start_serial)/CLOCKS_PER_SEC);

 
}